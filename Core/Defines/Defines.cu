#include "hip/hip_runtime.h"
#include <cmath>
#include "VectorCUDA.h"

namespace SingleNet
{
    namespace Defines
    {
        __device__ float SigmoidCUDA(float x)
        {
            return 1.0f / (1.0f + std::exp(-x));
        };

        __device__ float SigmoidDerivativeCUDA(float x)
        {
            return SigmoidCUDA(x) * (1.0f - SigmoidCUDA(x));
        };

        __device__ float SigmoidDerivativeCUDA_(float x)
        {
            return x * (1.0f - x);
        };

        __device__ float TanhCUDA(float x)
        {
            return std::tanh(x);
        };

        __device__ float TanhDerivativeCUDA(float x)
        {
            return 1.0f - std::pow(TanhCUDA(x), 2);
        };

        __device__ float TanhDerivativeCUDA_(float x)
        {
            return 1.0f - x * x;
        };

        __device__ float ReLUCUDA(float x)
        {
            return x > 0.0f ? x : 0.0f;
        };

        __device__ float ReLUDerivativeCUDA(float x)
        {
            return x > 0.0f ? 1.0f : 0.0f;
        };
    }
}