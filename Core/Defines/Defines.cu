#include "hip/hip_runtime.h"
#include <cmath>
#include "Core/Defines/Defines.cuh"

namespace SingleNet
{
    namespace Defines
    {
        // ------------------------------------------------------------
        // Activation functions
        // ------------------------------------------------------------

        // Sigmoid function
        std::function<float(float)> Sigmoid = [](float x)
        { return 1.0f / (1.0f + std::exp(-x)); };
        std::function<float(float)> SigmoidDerivative = [](float x)
        { return Sigmoid(x) * (1.0f - Sigmoid(x)); };
        std::function<float(float)> SigmoidDerivative_ = [](float x)
        { return x * (1.0f - x); };

        // Tanh function
        std::function<float(float)> Tanh = [](float x)
        { return std::tanh(x); };
        std::function<float(float)> TanhDerivative = [](float x)
        { return 1.0f - std::pow(Tanh(x), 2); };
        std::function<float(float)> TanhDerivative_ = [](float x)
        { return 1.0f - x * x; };

        // ReLU function
        std::function<float(float)> ReLU = [](float x)
        { return x > 0.0f ? x : 0.0f; };
        std::function<float(float)> ReLUDerivative = [](float x)
        { return x > 0.0f ? 1.0f : 0.0f; };

        // Softmax function
        std::function<Vector<float, 1>(Vector<float, 1>)> Softmax = [](Vector<float, 1> x)
        {
            float max = x[maxIndex(x)];
            float sum = 0.0f;

            for (size_t i = 0; i < x.size(); i++)
            {
                x[i] = std::exp(x[i] - max);
                sum += x[i];
            }

            for (size_t i = 0; i < x.size(); i++)
                x[i] /= sum;

            return x;
        };

        // ------------------------------------------------------------
        // Loss functions
        // ------------------------------------------------------------

        // Mean squared error
        std::function<float(Vector<float, 1>, Vector<float, 1>)> MSE = [](Vector<float, 1> y, Vector<float, 1> y_)
        {
            float sum = 0.0f;

            for (size_t i = 0; i < y.size(); i++)
                sum += std::pow(y[i] - y_[i], 2);

            return sum / y.size();
        };

        // Cross-entropy
        std::function<float(Vector<float, 1>, Vector<float, 1>)> CrossEntropy = [](Vector<float, 1> y, Vector<float, 1> y_)
        {
            float sum = 0.0f;

            for (size_t i = 0; i < y.size(); i++)
                if(y[i]) sum -= std::log(y_[i]);

            return sum;
        };

        __device__ float SigmoidCUDA(float x)
        {
            return 1.0f / (1.0f + std::exp(-x));
        };

        __device__ float SigmoidDerivativeCUDA(float x)
        {
            return SigmoidCUDA(x) * (1.0f - SigmoidCUDA(x));
        };

        __device__ float SigmoidDerivativeCUDA_(float x)
        {
            return x * (1.0f - x);
        };

        __device__ float TanhCUDA(float x)
        {
            return std::tanh(x);
        };

        __device__ float TanhDerivativeCUDA(float x)
        {
            return 1.0f - std::pow(TanhCUDA(x), 2);
        };

        __device__ float TanhDerivativeCUDA_(float x)
        {
            return 1.0f - x * x;
        };

        __device__ float ReLUCUDA(float x)
        {
            return x > 0.0f ? x : 0.0f;
        };

        __device__ float ReLUDerivativeCUDA(float x)
        {
            return x > 0.0f ? 1.0f : 0.0f;
        };
    }
}