#include "hip/hip_runtime.h"
#include "VectorCUDA.h"

hipError_t cuda_check(hipError_t err, int line, const char *file, bool abort = true)
{
    if (err != hipSuccess)
    {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err) + " at " + file + ":" + std::to_string(line));
    }
    return err;
}

#define CUDA_CHECK(err) cuda_check((err), __LINE__, __FILE__)
#define MAX_THREADS_PER_BLOCK 1024

__device__ void CUDA_GetDim(size_t required_size, size_t *block, size_t *threads)
{
    *threads = (required_size > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : required_size;
    *block = (required_size / *threads) + 1;
}

template <size_t N>
__global__ void CUDA_Add(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v3))[index] = (static_cast<float *>(v1))[index] + (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Add<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            (static_cast<void **>(v3))[index], shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_AddAssign(void *v1, void *v2, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v1))[index] += (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_AddAssign<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index], shape_reversed);
    }
}

__global__ void CUDA_Add_bias(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[1] <= index)
        return;

    size_t block, thread;
    CUDA_GetDim(shape_reversed[0], &block, &thread);
    CUDA_Add<1><<<block, thread>>>(
        (static_cast<void **>(v1))[index], v2,
        (static_cast<void **>(v3))[index], shape_reversed);
}

__global__ void CUDA_Add_bias_assign(void *v1, void *v2, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[1] <= index)
        return;

    size_t block, thread;
    CUDA_GetDim(shape_reversed[0], &block, &thread);
    CUDA_AddAssign<1><<<block, thread>>>(
        (static_cast<void **>(v1))[index], v2, shape_reversed);
}

template <size_t N>
__global__ void CUDA_Sub(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v3))[index] = (static_cast<float *>(v1))[index] - (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Sub<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            (static_cast<void **>(v3))[index], shape_reversed);
    }
}

__global__ void CUDA_Sub_bias(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[1] <= index)
        return;

    size_t block, thread;
    CUDA_GetDim(shape_reversed[0], &block, &thread);
    CUDA_Sub<1><<<block, thread>>>(
        (static_cast<void **>(v1))[index], v2,
        (static_cast<void **>(v3))[index], shape_reversed);
}

template <size_t N>
__global__ void CUDA_SubAssign(void *v1, void *v2, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v1))[index] -= (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_SubAssign<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index], shape_reversed);
    }
}

__global__ void CUDA_Sub_bias_assign(void *v1, void *v2, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[1] <= index)
        return;

    size_t block, thread;
    CUDA_GetDim(shape_reversed[0], &block, &thread);
    CUDA_SubAssign<1><<<block, thread>>>(
        (static_cast<void **>(v1))[index], v2, shape_reversed);
}

template <size_t N>
__global__ void CUDA_Mul(void *src, void *dst, float value, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(dst))[index] = (static_cast<float *>(src))[index] * value;
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Mul<N - 1><<<block, thread>>>(
            (static_cast<void **>(src))[index],
            (static_cast<void **>(dst))[index], value, shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_MulAssign(void *v1, float value, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v1))[index] *= value;
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_MulAssign<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index], value, shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_Times(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v3))[index] = (static_cast<float *>(v1))[index] * (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Times<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            (static_cast<void **>(v3))[index], shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_Copy(void *v1, void *v2, size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v2))[index] = (static_cast<float *>(v1))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Copy<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index], shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_Map(void *v1, void *v2, float (*func)(float), size_t *shape_reversed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v2))[index] = func((static_cast<float *>(v1))[index]);
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Map<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            func, shape_reversed);
    }
}

__global__ void CUDA_Transpose_helper(void *v1, void *v2, size_t *shape, size_t index_previous)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape[1] <= index)
        return;

    (static_cast<float **>(v2))[index][index_previous] = (static_cast<float **>(v1))[index_previous][index];
}

__global__ void CUDA_Transpose(void *v1, void *v2, size_t *shape)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (shape[0] <= index)
        return;

    size_t block, thread;
    CUDA_GetDim(shape[1], &block, &thread);
    CUDA_Transpose_helper<<<block, thread>>>(v1, v2, shape, index);
}

__global__ void CUDA_Dot_1D(void *v1, void *v2, void *result, size_t size)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (size <= index)
        return;

    float temp = (static_cast<float *>(v1))[index] * (static_cast<float *>(v2))[index];
    atomicAdd(static_cast<float *>(result), temp);
}

__global__ void CUDA_Dot_2D1D(void *v1, void *v2, void *result, size_t size, size_t v1_size)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (index < v1_size)
    {
        size_t block, thread;
        CUDA_GetDim(size, &block, &thread);
        CUDA_Dot_1D<<<block, thread>>>(((void **)v1)[index], v2, &((float *)result)[index], size);
    }
}

__global__ void CUDA_Dot_2D(void *v1, void *v2, void *result, size_t size, size_t v1_size, size_t v2_size)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (index < v1_size)
    {
        size_t block, thread;
        CUDA_GetDim(v2_size, &block, &thread);
        CUDA_Dot_2D1D<<<block, thread>>>(v2, ((void **)v1)[index], ((void **)result)[index], size, v2_size);
    }
}

__global__ void CUDA_Sum_1D(void *v1, void *result, size_t size)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (size <= index)
        return;

    atomicAdd(static_cast<float *>(result), (static_cast<float *>(v1))[index]);
}

__global__ void CUDA_Sum_2D1D(void *v1, void *result, size_t size, size_t v1_size)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    if (index < v1_size)
    {
        size_t block, thread;
        CUDA_GetDim(size, &block, &thread);
        CUDA_Sum_1D<<<block, thread>>>(((void **)v1)[index], &((float *)result)[index], size);
    }
}

namespace SingleNet
{
    template <size_t N>
    void *malloc_device(const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            void *dst = nullptr;
            CUDA_CHECK(hipMalloc(&dst, sizeof(float) * shape_reversed[0]));
            return dst;
        }
        else
        {
            void *dst_device, **dst_host;

            CUDA_CHECK(hipMalloc(&dst_device, shape_reversed[N - 1] * sizeof(void *)));
            dst_host = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                dst_host[i] = malloc_device<N - 1>(shape_reversed);
            }

            CUDA_CHECK(hipMemcpy(dst_device, dst_host, shape_reversed[N - 1] * sizeof(void *), hipMemcpyHostToDevice));
            free(dst_host);

            return dst_device;
        }

        return nullptr;
    }

    template <size_t N>
    void free_device(void *device_pointer, const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            CUDA_CHECK(hipFree(device_pointer));
        }
        else
        {
            void **host_pointer = (void **)malloc(sizeof(void *) * shape_reversed[N - 1]);
            CUDA_CHECK(hipMemcpy(host_pointer, device_pointer, sizeof(void *) * shape_reversed[N - 1], hipMemcpyDeviceToHost));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                free_device<N - 1>(host_pointer[i], shape_reversed);
            }

            CUDA_CHECK(hipFree(device_pointer));
            free(host_pointer);
        }
    }

    template <size_t N>
    void *memcpy_device(const Vector<size_t, 1> &shape_reversed, const Vector<float, N> &v)
    {
        if constexpr (N == 1)
        {
            void *dst = nullptr;
            CUDA_CHECK(hipMalloc(&dst, sizeof(float) * shape_reversed[0]));
            CUDA_CHECK(hipMemcpy(dst, v.data(), sizeof(float) * shape_reversed[0], hipMemcpyHostToDevice));
            return dst;
        }
        else
        {
            void *dst_device, **dst_host;

            CUDA_CHECK(hipMalloc(&dst_device, shape_reversed[N - 1] * sizeof(void *)));
            dst_host = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                dst_host[i] = memcpy_device<N - 1>(shape_reversed, v[i]);
            }

            CUDA_CHECK(hipMemcpy(dst_device, dst_host, shape_reversed[N - 1] * sizeof(void *), hipMemcpyHostToDevice));
            free(dst_host);

            return dst_device;
        }

        return nullptr;
    }

    template <size_t N>
    Vector<float, N> memcpy_host(const Vector<size_t, 1> &shape_reversed, void *device_pointer)
    {
        if constexpr (N == 1)
        {
            Vector<float, 1> v(shape_reversed[0]);
            CUDA_CHECK(hipMemcpy(v.data(), device_pointer, sizeof(float) * shape_reversed[0], hipMemcpyDeviceToHost));
            return v;
        }
        else
        {
            void **host_pointer = (void **)malloc(sizeof(void *) * shape_reversed[N - 1]);
            CUDA_CHECK(hipMemcpy(host_pointer, device_pointer, sizeof(void *) * shape_reversed[N - 1], hipMemcpyDeviceToHost));

            Vector<float, N> v(shape_reversed[N - 1]);
            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                v[i] = memcpy_host<N - 1>(shape_reversed, host_pointer[i]);
            }

            free(host_pointer);
            return v;
        }

        return Vector<float, N>();
    }

    template <size_t N>
    VectorCUDA<N>::VectorCUDA()
    {
        this->m_pDeviceData = nullptr;
        this->m_shape = Vector<size_t, 1>();
    }

    template <size_t N>
    VectorCUDA<N>::VectorCUDA(const Vector<float, N> &v)
    {
        this->m_pDeviceData = memcpy_device(reverse(shape(v)), v);
        this->m_shape = shape(v);
    }

    template <size_t N>
    VectorCUDA<N>::VectorCUDA(const VectorCUDA<N> &v)
    {
        *this = v.copy();
    }

    template <size_t N>
    VectorCUDA<N>::VectorCUDA(VectorCUDA<N> &v)
    {
        this->m_pDeviceData = v.m_pDeviceData;
        this->m_shape = v.m_shape;

        v.m_pDeviceData = nullptr;
        v.m_shape = Vector<size_t, 1>();
    }

    template <size_t N>
    VectorCUDA<N>::~VectorCUDA()
    {
        if (this->m_pDeviceData)
            free_device<N>(this->m_pDeviceData, (this->m_shape));
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator=(const Vector<float, N> &v)
    {
        if (this->m_pDeviceData)
            free_device<N>(this->m_pDeviceData, (this->m_shape));

        this->m_pDeviceData = memcpy_device(reverse(shape(v)), v);
        this->m_shape = shape(v);

        return *this;
    }
    
    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator=(const VectorCUDA<N> &v)
    {
        *this = v.copy();
        return *this;
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator=(VectorCUDA<N> &v)
    {
        if (this->m_pDeviceData)
            free_device<N>(this->m_pDeviceData, (this->m_shape));

        this->m_pDeviceData = v.m_pDeviceData;
        this->m_shape = v.m_shape;

        v.m_pDeviceData = nullptr;
        v.m_shape = Vector<size_t, 1>();

        return *this;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::operator+(const VectorCUDA<N> &v) const
    {
        if (this->m_shape != v.m_shape)
        {
            throw std::runtime_error("VectorCUDA<N>::operator+: shape not match");
        }

        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(reverse(this->m_shape));
        dst.m_shape = this->m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_Add<N><<<blocks, threads>>>(this->m_pDeviceData, v.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::operator-(const VectorCUDA<N> &v) const
    {
        if (this->m_shape != v.m_shape)
        {
            throw std::runtime_error("VectorCUDA<N>::operator-: shape not match");
        }

        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(reverse(this->m_shape));
        dst.m_shape = this->m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_Sub<N><<<blocks, threads>>>(this->m_pDeviceData, v.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator+=(const VectorCUDA<N> &v)
    {
        if (this->m_shape != v.m_shape)
        {
            throw std::runtime_error("VectorCUDA<N>::operator+=: shape not match");
        }

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_AddAssign<N><<<blocks, threads>>>(this->m_pDeviceData, v.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return *this;
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator-=(const VectorCUDA<N> &v)
    {
        if (this->m_shape != v.m_shape)
        {
            throw std::runtime_error("VectorCUDA<N>::operator-=: shape not match");
        }

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_SubAssign<N><<<blocks, threads>>>(this->m_pDeviceData, v.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return *this;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::operator*(const float &s) const
    {
        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(reverse(this->m_shape));
        dst.m_shape = this->m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_Mul<N><<<blocks, threads>>>(this->m_pDeviceData, dst.m_pDeviceData, s, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::operator/(const float &s) const
    {
        if (s == 0)
        {
            throw std::runtime_error("VectorCUDA<N>::operator/: divisor is zero");
        }
        return (*this * (1.0f / s));
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator*=(const float &s)
    {
        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_MulAssign<N><<<blocks, threads>>>(this->m_pDeviceData, s, shape_ptr_device);

        hipFree(shape_ptr_device);
        return *this;
    }

    template <size_t N>
    VectorCUDA<N> &VectorCUDA<N>::operator/=(const float &s)
    {
        if (s == 0)
        {
            throw std::runtime_error("VectorCUDA<N>::operator/=: divisor is zero");
        }

        *this *= (1.0f / s);
        return *this;
    }

    VectorCUDA<2> operator+(const VectorCUDA<2> &v1, const VectorCUDA<1> &v2)
    {
        if (v1.m_shape[1] != v2.m_shape[0])
        {
            throw std::runtime_error("VectorCUDA<2>::operator+: shape not match");
        }

        VectorCUDA<2> dst;
        dst.m_pDeviceData = malloc_device<2>(v1.m_shape);
        dst.m_shape = v1.m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * 2);
        hipMemcpy(shape_ptr_device, reverse(v1.m_shape).data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Add_bias<<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return dst;
    }

    VectorCUDA<2> operator-(const VectorCUDA<2> &v1, const VectorCUDA<1> &v2)
    {
        if (v1.m_shape[1] != v2.m_shape[0])
        {
            throw std::runtime_error("VectorCUDA<2>::operator-: shape not match");
        }

        VectorCUDA<2> dst;
        dst.m_pDeviceData = malloc_device<2>(v1.m_shape);
        dst.m_shape = v1.m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * 2);
        hipMemcpy(shape_ptr_device, reverse(v1.m_shape).data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Sub_bias<<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return dst;
    }

    VectorCUDA<2> &operator+=(VectorCUDA<2> &v1, const VectorCUDA<1> &v2)
    {
        if (v1.m_shape[1] != v2.m_shape[0])
        {
            throw std::runtime_error("VectorCUDA<2>::operator+=: shape not match");
        }

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * 2);
        hipMemcpy(shape_ptr_device, reverse(v1.m_shape).data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Add_bias_assign<<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return v1;
    }

    VectorCUDA<2> &operator-=(VectorCUDA<2> &v1, const VectorCUDA<1> &v2)
    {
        if (v1.m_shape[1] != v2.m_shape[0])
        {
            throw std::runtime_error("VectorCUDA<2>::operator-=: shape not match");
        }

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * 2);
        hipMemcpy(shape_ptr_device, reverse(v1.m_shape).data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Sub_bias_assign<<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, shape_ptr_device);
        hipFree(shape_ptr_device);

        return v1;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::map(float (*func)(float)) const
    {
        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(reverse(this->m_shape));
        dst.m_shape = this->m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_Map<N><<<blocks, threads>>>(this->m_pDeviceData, dst.m_pDeviceData, func, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    template <size_t N>
    VectorCUDA<N> VectorCUDA<N>::copy() const
    {
        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(reverse(this->m_shape));
        dst.m_shape = this->m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(this->m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (m_shape[0] < MAX_THREADS_PER_BLOCK) ? m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (m_shape[0] / threads) + 1;

        CUDA_Copy<N><<<blocks, threads>>>(this->m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    VectorCUDA<1> mean(const VectorCUDA<2> &v)
    {
        VectorCUDA<1> dst;
        dst.m_pDeviceData = malloc_device<1>({v.m_shape[1]});
        dst.m_shape = {v.m_shape[1]};

        size_t threads = (v.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v.m_shape[0] / threads) + 1;

        CUDA_Sum_2D1D<<<blocks, threads>>>(v.m_pDeviceData, dst.m_pDeviceData, v.m_shape[1], v.m_shape[0]);
        return dst;
    }

    VectorCUDA<2> transpose(const VectorCUDA<2> &v)
    {
        VectorCUDA<2> dst;
        dst.m_pDeviceData = malloc_device<2>(v.m_shape);
        dst.m_shape = reverse(v.m_shape);

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * 2);
        hipMemcpy(shape_ptr_device, v.m_shape.data(), sizeof(size_t) * 2, hipMemcpyHostToDevice);

        size_t threads = (v.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v.m_shape[0] / threads) + 1;

        CUDA_Transpose<<<blocks, threads>>>(v.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    VectorCUDA<2> dot(const VectorCUDA<2> &v1, const VectorCUDA<2> &v2)
    {
        if (v1.m_shape[1] != v2.m_shape[0])
        {
            throw std::runtime_error("VectorCUDA<2>::dot: shape not match");
        }

        VectorCUDA<2> dst;
        dst.m_pDeviceData = malloc_device<2>({v2.m_shape[1], v1.m_shape[0]});
        dst.m_shape = {v1.m_shape[0], v2.m_shape[1]};

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Dot_2D<<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, dst.m_pDeviceData, v1.m_shape[1], v1.m_shape[0], v2.m_shape[1]);

        return dst;
    }

    template <size_t N>
    VectorCUDA<N> times(const VectorCUDA<N> &v1, const VectorCUDA<N> &v2)
    {
        if (v1.m_shape != v2.m_shape)
        {
            throw std::runtime_error("VectorCUDA<N>::times: shape not match");
        }

        VectorCUDA<N> dst;
        dst.m_pDeviceData = malloc_device<N>(v1.m_shape);
        dst.m_shape = v1.m_shape;

        size_t *shape_ptr_device;
        hipMalloc(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, reverse(v1.m_shape).data(), sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t threads = (v1.m_shape[0] < MAX_THREADS_PER_BLOCK) ? v1.m_shape[0] : MAX_THREADS_PER_BLOCK;
        size_t blocks = (v1.m_shape[0] / threads) + 1;

        CUDA_Times<N><<<blocks, threads>>>(v1.m_pDeviceData, v2.m_pDeviceData, dst.m_pDeviceData, shape_ptr_device);

        hipFree(shape_ptr_device);
        return dst;
    }

    template VectorCUDA<3> times(const VectorCUDA<3> &v1, const VectorCUDA<3> &v2);
    template VectorCUDA<2> times(const VectorCUDA<2> &v1, const VectorCUDA<2> &v2);
    template VectorCUDA<1> times(const VectorCUDA<1> &v1, const VectorCUDA<1> &v2);

    template <size_t N>
    Vector<float, N> to_cpu(const VectorCUDA<N> &v)
    {
        return memcpy_host<N>(reverse(v.m_shape), v.m_pDeviceData);
    }

    template Vector<float, 3> to_cpu(const VectorCUDA<3> &v);
    template Vector<float, 2> to_cpu(const VectorCUDA<2> &v);
    template Vector<float, 1> to_cpu(const VectorCUDA<1> &v);
}

template class SingleNet::VectorCUDA<3>;
template class SingleNet::VectorCUDA<2>;
template class SingleNet::VectorCUDA<1>;
