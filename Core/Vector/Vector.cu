#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <string>
#include <stdexcept>

#include "Vector.h"

#define MAX_BLOCKS 256

hipError_t cuda_check(hipError_t err, int line, const char *file, bool abort = true)
{
    if (err != hipSuccess)
    {
        throw std::runtime_error(std::string("CUDA error: ") + hipGetErrorString(err) + " at " + file + ":" + std::to_string(line));
    }
    return err;
}

#define CUDA_CHECK(err) cuda_check((err), __LINE__, __FILE__)

__device__ void CUDA_GetDim(size_t required_size, size_t *block, size_t *threads)
{
    *block = MAX_BLOCKS;
    *threads = required_size / MAX_BLOCKS + 1;
}

template <size_t N>
__global__ void CUDA_Add(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v3))[index] = (static_cast<float *>(v1))[index] + (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Add<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            (static_cast<void **>(v3))[index], shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_Sub(void *v1, void *v2, void *v3, size_t *shape_reversed)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v3))[index] = (static_cast<float *>(v1))[index] - (static_cast<float *>(v2))[index];
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Sub<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index],
            (static_cast<void **>(v2))[index],
            (static_cast<void **>(v3))[index], shape_reversed);
    }
}

template <size_t N>
__global__ void CUDA_Mul(void *v1, float *value, size_t *shape_reversed)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (shape_reversed[N - 1] <= index)
        return;

    if constexpr (N == 1)
    {
        (static_cast<float *>(v1))[index] *= *value;
    }
    else
    {
        size_t block, thread;
        CUDA_GetDim(shape_reversed[N - 2], &block, &thread);
        CUDA_Mul<N - 1><<<block, thread>>>(
            (static_cast<void **>(v1))[index], value, shape_reversed);
    }
}

__global__ void CUDA_Dot_1D(void *v1, void *v2, void *result, size_t size)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (size <= index)
        return;

    float temp = (static_cast<float *>(v1))[index] * (static_cast<float *>(v2))[index];
    atomicAdd(static_cast<float *>(result), temp);
}

__global__ void CUDA_Dot_2D1D(void *v1, void *v2, void *result, size_t size, size_t v1_size)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (index < v1_size)
    {
        size_t block, thread;
        CUDA_GetDim(size, &block, &thread);
        CUDA_Dot_1D<<<block, thread>>>(((void **)v1)[index], v2, &((float *)result)[index], size);
    }
}

__global__ void CUDA_Dot_2D(void *v1, void *v2, void *result, size_t size, size_t v1_size, size_t v2_size)
{
    int index = MAX_BLOCKS * threadIdx.x + blockIdx.x;

    if (index < v1_size) 
    {
        size_t block, thread;
        CUDA_GetDim(v2_size, &block, &thread);
        CUDA_Dot_2D1D<<<block, thread>>>(v2, ((void **)v1)[index], ((void **)result)[index], size, v2_size);
    }
}

namespace SingleNet
{

    template <size_t N>
    void *CUDA_Memcpy(void *src, const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            void *dst = nullptr;

            CUDA_CHECK(hipMallocManaged(&dst, sizeof(float) * shape_reversed[0]));
            CUDA_CHECK(hipMemcpy(dst, src, sizeof(float) * shape_reversed[0], hipMemcpyHostToDevice));

            return dst;
        }
        else
        {
            void *dst_device, **dst_host;

            CUDA_CHECK(hipMallocManaged(&dst_device, shape_reversed[N - 1] * sizeof(void *)));
            dst_host = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                dst_host[i] = CUDA_Memcpy<N - 1>((static_cast<void **>(src))[i], shape_reversed);
            }

            CUDA_CHECK(hipMemcpy(dst_device, dst_host, shape_reversed[N - 1] * sizeof(void *), hipMemcpyHostToDevice));
            free(dst_host);

            return dst_device;
        }

        return nullptr;
    }

    template <size_t N>
    void *CUDA_Malloc(const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            void *dst = nullptr;
            CUDA_CHECK(hipMallocManaged(&dst, sizeof(float) * shape_reversed[0]));
            return dst;
        }
        else
        {
            void *dst_device, **dst_host;

            CUDA_CHECK(hipMallocManaged(&dst_device, shape_reversed[N - 1] * sizeof(void *)));
            dst_host = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                dst_host[i] = CUDA_Malloc<N - 1>(shape_reversed);
            }

            CUDA_CHECK(hipMemcpy(dst_device, dst_host, shape_reversed[N - 1] * sizeof(void *), hipMemcpyHostToDevice));
            free(dst_host);

            return dst_device;
        }

        return nullptr;
    }

    template <size_t N>
    void CUDA_Free(void *device_pointer, const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            CUDA_CHECK(hipFree(device_pointer));
        }
        else
        {
            void **host_pointer = (void **)malloc(sizeof(void *) * shape_reversed[N - 1]);
            CUDA_CHECK(hipMemcpy(host_pointer, device_pointer, sizeof(void *) * shape_reversed[N - 1], hipMemcpyDeviceToHost));

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                CUDA_Free<N - 1>(host_pointer[i], shape_reversed);
            }

            CUDA_CHECK(hipFree(device_pointer));
            free(host_pointer);
        }
    }

    template <size_t N>
    void *Host_Memcpy(void *device_src, const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            void *dst = malloc(sizeof(float) * shape_reversed[0]);
            CUDA_CHECK(hipMemcpy(dst, device_src, sizeof(float) * shape_reversed[0], hipMemcpyDeviceToHost));
            return dst;
        }
        else
        {
            void **device_src_copy = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));
            void **dst_host = (void **)malloc(shape_reversed[N - 1] * sizeof(void *));

            hipError_t err = hipMemcpy(device_src_copy, device_src, shape_reversed[N - 1] * sizeof(void *), hipMemcpyDeviceToHost);
            CUDA_CHECK(err);

            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                dst_host[i] = Host_Memcpy<N - 1>(device_src_copy[i], shape_reversed);
            }

            free(device_src_copy);
            return dst_host;
        }

        return nullptr;
    }

    template <size_t N>
    void Host_Free(void *ptr, const Vector<size_t, 1> &shape_reversed)
    {
        if constexpr (N == 1)
        {
            free(ptr);
        }
        else
        {
            for (size_t i = 0; i < shape_reversed[N - 1]; i++)
            {
                Host_Free<N - 1>(
                    (static_cast<void **>(ptr))[i],
                    shape_reversed);
            }

            free(ptr);
        }
    }

    template <size_t N>
    Vector<float, N> operator+(const Vector<float, N> &v1, const Vector<float, N> &v2)
    {
        if (shape(v1) != shape(v2))
            throw std::runtime_error("Vector<float>::Shape mismatch");

        void *v1_ptr = to_pointer(v1);
        void *v2_ptr = to_pointer(v2);

        void *v1_ptr_device = CUDA_Memcpy<N>(v1_ptr, reverse(shape(v1)));
        void *v2_ptr_device = CUDA_Memcpy<N>(v2_ptr, reverse(shape(v2)));

        void *v3_ptr_device = CUDA_Malloc<N>(reverse(shape(v1)));

        void *shape_ptr = to_pointer(reverse(shape(v1)));
        size_t *shape_ptr_device;

        hipMallocManaged(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, shape_ptr, sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t block = MAX_BLOCKS;
        size_t threads = (v1.size() / MAX_BLOCKS) + 1;

        CUDA_Add<N><<<block, threads>>>(v1_ptr_device, v2_ptr_device, v3_ptr_device, shape_ptr_device);

        void *v3_ptr = Host_Memcpy<N>(v3_ptr_device, reverse(shape(v1)));

        CUDA_Free<N>(v1_ptr_device, reverse(shape(v1)));
        CUDA_Free<N>(v2_ptr_device, reverse(shape(v2)));
        CUDA_Free<N>(v3_ptr_device, reverse(shape(v1)));
        hipFree(shape_ptr_device);

        Vector<float, N> result = from_pointer<float, N>(v3_ptr, reverse(shape(v1)));
        return result;
    }

    template Vector<float, 3> operator+(const Vector<float, 3> &v1, const Vector<float, 3> &v2);
    template Vector<float, 2> operator+(const Vector<float, 2> &v1, const Vector<float, 2> &v2);
    template Vector<float, 1> operator+(const Vector<float, 1> &v1, const Vector<float, 1> &v2);

    template <size_t N>
    Vector<float, N> operator-(const Vector<float, N> &v1, const Vector<float, N> &v2)
    {
        if (shape(v1) != shape(v2))
            throw std::runtime_error("Vector<float>::Shape mismatch");

        void *v1_ptr = to_pointer(v1);
        void *v2_ptr = to_pointer(v2);

        void *v1_ptr_device = CUDA_Memcpy<N>(v1_ptr, reverse(shape(v1)));
        void *v2_ptr_device = CUDA_Memcpy<N>(v2_ptr, reverse(shape(v2)));

        void *v3_ptr_device = CUDA_Malloc<N>(reverse(shape(v1)));

        void *shape_ptr = to_pointer(reverse(shape(v1)));
        size_t *shape_ptr_device;

        hipMallocManaged(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, shape_ptr, sizeof(size_t) * N, hipMemcpyHostToDevice);

        size_t block = MAX_BLOCKS;
        size_t threads = (v1.size() / MAX_BLOCKS) + 1;

        CUDA_Sub<N><<<block, threads>>>(v1_ptr_device, v2_ptr_device, v3_ptr_device, shape_ptr_device);

        void *v3_ptr = Host_Memcpy<N>(v3_ptr_device, reverse(shape(v1)));

        CUDA_Free<N>(v1_ptr_device, reverse(shape(v1)));
        CUDA_Free<N>(v2_ptr_device, reverse(shape(v2)));
        CUDA_Free<N>(v3_ptr_device, reverse(shape(v1)));
        hipFree(shape_ptr_device);

        Vector<float, N> result = from_pointer<float, N>(v3_ptr, reverse(shape(v1)));
        return result;
    }

    template Vector<float, 3> operator-(const Vector<float, 3> &v1, const Vector<float, 3> &v2);
    template Vector<float, 2> operator-(const Vector<float, 2> &v1, const Vector<float, 2> &v2);
    template Vector<float, 1> operator-(const Vector<float, 1> &v1, const Vector<float, 1> &v2);

    template <size_t N>
    Vector<float, N> &operator+=(Vector<float, N> &v1, const Vector<float, N> &v2)
    {
        v1 = v1 + v2;
        return v1;
    }

    template Vector<float, 3> &operator+=(Vector<float, 3> &v1, const Vector<float, 3> &v2);
    template Vector<float, 2> &operator+=(Vector<float, 2> &v1, const Vector<float, 2> &v2);
    template Vector<float, 1> &operator+=(Vector<float, 1> &v1, const Vector<float, 1> &v2);

    template <size_t N>
    Vector<float, N> &operator-=(Vector<float, N> &v1, const Vector<float, N> &v2)
    {
        v1 = v1 - v2;
        return v1;
    }

    template Vector<float, 3> &operator-=(Vector<float, 3> &v1, const Vector<float, 3> &v2);
    template Vector<float, 2> &operator-=(Vector<float, 2> &v1, const Vector<float, 2> &v2);
    template Vector<float, 1> &operator-=(Vector<float, 1> &v1, const Vector<float, 1> &v2);

    template <size_t N>
    Vector<float, N> operator*(const Vector<float, N> &v1, const float &s)
    {
        void *v1_ptr = to_pointer(v1);
        void *v1_ptr_device = CUDA_Memcpy<N>(v1_ptr, reverse(shape(v1)));

        void *shape_ptr = to_pointer(reverse(shape(v1)));
        size_t *shape_ptr_device;

        hipMallocManaged(&shape_ptr_device, sizeof(size_t) * N);
        hipMemcpy(shape_ptr_device, shape_ptr, sizeof(size_t) * N, hipMemcpyHostToDevice);

        float *s_ptr_device;
        hipMallocManaged(&s_ptr_device, sizeof(float));
        hipMemcpy(s_ptr_device, &s, sizeof(float), hipMemcpyHostToDevice);

        size_t block = MAX_BLOCKS;
        size_t threads = (v1.size() / MAX_BLOCKS) + 1;

        CUDA_Mul<N><<<block, threads>>>(v1_ptr_device, s_ptr_device, shape_ptr_device);

        void *result_ptr = Host_Memcpy<N>(v1_ptr_device, reverse(shape(v1)));

        CUDA_Free<N>(v1_ptr_device, reverse(shape(v1)));
        hipFree(shape_ptr_device);
        hipFree(s_ptr_device);

        Vector<float, N> result = from_pointer<float, N>(result_ptr, reverse(shape(v1)));
        return result;
    }

    template Vector<float, 3> operator*(const Vector<float, 3> &v1, const float &s);
    template Vector<float, 2> operator*(const Vector<float, 2> &v1, const float &s);
    template Vector<float, 1> operator*(const Vector<float, 1> &v1, const float &s);

    template <size_t N>
    Vector<float, N> &operator*=(Vector<float, N> &v1, const float &s)
    {
        v1 = v1 * s;
        return v1;
    }

    template Vector<float, 3> &operator*=(Vector<float, 3> &v1, const float &s);
    template Vector<float, 2> &operator*=(Vector<float, 2> &v1, const float &s);
    template Vector<float, 1> &operator*=(Vector<float, 1> &v1, const float &s);

    template <size_t N>
    Vector<float, N> operator/(const Vector<float, N> &v1, const float &s)
    {
        return v1 * (1.0f / s);
    }

    template Vector<float, 3> operator/(const Vector<float, 3> &v1, const float &s);
    template Vector<float, 2> operator/(const Vector<float, 2> &v1, const float &s);
    template Vector<float, 1> operator/(const Vector<float, 1> &v1, const float &s);

    template <size_t N>
    Vector<float, N> &operator/=(Vector<float, N> &v1, const float &s)
    {
        v1 = v1 * (1.0f / s);
        return v1;
    }

    template Vector<float, 3> &operator/=(Vector<float, 3> &v1, const float &s);
    template Vector<float, 2> &operator/=(Vector<float, 2> &v1, const float &s);
    template Vector<float, 1> &operator/=(Vector<float, 1> &v1, const float &s);

    float dot(const Vector<float, 1> &v1, const Vector<float, 1> &v2)
    {
        if (shape(v1) != shape(v2))
        {
            throw std::runtime_error("dot product error: shape mismatch");
        }

        void *v1_ptr = to_pointer(v1);
        void *v2_ptr = to_pointer(v2);

        void *v1_ptr_device = CUDA_Memcpy<1>(v1_ptr, shape(v1));
        void *v2_ptr_device = CUDA_Memcpy<1>(v2_ptr, shape(v2));

        void *result_ptr_device = nullptr;
        hipMallocManaged(&result_ptr_device, sizeof(float));

        size_t block = MAX_BLOCKS;
        size_t threads = (v1.size() / MAX_BLOCKS) + 1;

        CUDA_Dot_1D<<<block, threads>>>(v1_ptr_device, v2_ptr_device, result_ptr_device, v1.size());
        hipError_t err = hipDeviceSynchronize();

        if (err != hipSuccess)
        {
            throw std::runtime_error("dot product error: " + std::string(hipGetErrorString(err)));
        }

        float result = 0.0f;
        hipMemcpy(&result, result_ptr_device, sizeof(float), hipMemcpyDeviceToHost);

        CUDA_Free<1>(v1_ptr_device, shape(v1));
        CUDA_Free<1>(v2_ptr_device, shape(v2));
        hipFree(result_ptr_device);

        return result;
    }

    Vector<float, 1> dot(const Vector<float, 2> &v1, const Vector<float, 1> &v2)
    {
        if (shape(v1)[1] != shape(v2)[0])
        {
            throw std::runtime_error("dot product error: shape mismatch");
        }

        try
        {
            void *v1_ptr = to_pointer(v1);
            void *v2_ptr = to_pointer(v2);

            void *v1_ptr_device = CUDA_Memcpy<2>(v1_ptr, reverse(shape(v1)));
            void *v2_ptr_device = CUDA_Memcpy<1>(v2_ptr, shape(v2));

            void *result_ptr_device;
            hipMallocManaged(&result_ptr_device, shape(v1)[0] * sizeof(float));

            size_t block = MAX_BLOCKS;
            size_t threads = (v1.size() / MAX_BLOCKS) + 1;

            CUDA_Dot_2D1D<<<block, threads>>>(v1_ptr_device, v2_ptr_device, result_ptr_device, shape(v1)[1], shape(v1)[0]);
            hipError_t err = hipDeviceSynchronize();

            if (err != hipSuccess)
            {
                throw std::runtime_error("dot product error: " + std::string(hipGetErrorString(err)));
            }

            void *result_ptr = malloc(shape(v1)[0] * sizeof(float));
            hipMemcpy(result_ptr, result_ptr_device, shape(v1)[0] * sizeof(float), hipMemcpyDeviceToHost);

            CUDA_Free<2>(v1_ptr_device, reverse(shape(v1)));
            CUDA_Free<1>(v2_ptr_device, shape(v2));
            hipFree(result_ptr_device);

            Vector<float, 1> result = from_pointer<float, 1>(result_ptr, {shape(v1)[0]});
            return result;
        }
        catch (std::exception &e)
        {
            throw std::runtime_error("Vector<float, 1> dot():\n\t" + std::string(e.what()));
        }
    }

    Vector<float, 2> dot(const Vector<float, 2> &v1, const Vector<float, 2> &v2)
    {
        if (shape(v1)[1] != shape(v2)[0])
        {
            throw std::runtime_error("dot product error: shape mismatch");
        }
        try {
            Vector<float, 2> v2_transposed = transpose(v2);

            void *v1_ptr = to_pointer(v1);
            void *v2_ptr = to_pointer(v2_transposed);

            void *v1_ptr_device = CUDA_Memcpy<2>(v1_ptr, reverse(shape(v1)));
            void *v2_ptr_device = CUDA_Memcpy<2>(v2_ptr, reverse(shape(v2_transposed)));

            void *result_ptr_device = CUDA_Malloc<2>({shape(v2)[1], shape(v1)[0]});

            size_t block = MAX_BLOCKS;
            size_t threads = (v1.size() / MAX_BLOCKS) + 1;

            CUDA_Dot_2D<<<block, threads>>>(v1_ptr_device, v2_ptr_device, result_ptr_device, shape(v1)[1], shape(v1)[0], shape(v2)[1]);
            hipError_t err = hipDeviceSynchronize();

            if (err != hipSuccess)
            {
                throw std::runtime_error("dot product error: " + std::string(hipGetErrorString(err)));
            }

            void *result_ptr = Host_Memcpy<2>(result_ptr_device, {shape(v2)[1], shape(v1)[0]});

            CUDA_Free<2>(v1_ptr_device, reverse(shape(v1)));
            CUDA_Free<2>(v2_ptr_device, shape(v2));
            CUDA_Free<2>(result_ptr_device, {shape(v2)[1], shape(v1)[0]});

            Vector<float, 2> result = from_pointer<float, 2>(result_ptr, {shape(v2)[1], shape(v1)[0]});
            return result;
        }
        catch (std::exception &e)
        {
            throw std::runtime_error("Vector<float, 2> dot():\n\t" + std::string(e.what()));
        }
    }
}